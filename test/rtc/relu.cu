#include "hip/hip_runtime.h"
extern "C"  __global__ void %(cu_func_name)( float * const out ) {
  int32_t const ix = blockDim.x * blockIdx.x + threadIdx.x;
  if( ix < %(out_sz) ) { out[ix] = (out[ix] <= 0) ? 0.0f : out[ix]; }
}
