#include "hip/hip_runtime.h"
extern "C"  __global__ void %(cu_func_name)( float const * const in, float * const out ) {
  int32_t const in_ix = blockDim.x * blockIdx.x + threadIdx.x;
  if( in_ix >= %(in_sz) ) { return; }
  out[in_ix] = in[in_ix];
}

/*

in_pels = num_img * in.sz.dims_prod()
num_in_blks = u32_ceil_div( in_pels, block_chan_pels )

normal in dims: img, chan, y, x OR img, chan, pels // where pels = x,y dims merged

block_iters = u32_ceil_div( chan, in_chan_tile ) // for ccp1, 96/8=12
pad_chan =  block_iter * in_chan_tile // pad by up to (in_chan_tile-1) [typ. 8; pad with zeros? garbage okay?]
block_chan_pels = t_tile_sz*tix_pels_tile_sz // typically 8*8=64
block_iter_pels = block_chan_pels * in_chan_tile; // typically 512

block_pels = 12*512 = 6144 // note: 24576 bytes, prob. too big for SM to fully cache, but 512=2K (per-iter cache) is fine.


xposed in dims (inner): (block_iter,  block_iter_chan, block_iter_pel)  == block_pel
            sz (inner): (block_iters, in_chan_tile,    block_chan_pels) == block_pels (only inner 2 dims need to be linear?)

*/
