#include "hip/hip_runtime.h"
// each thread: computes 8x8 block of out
// loop over k dim
extern "C"  __global__ void %(cu_func_name)( float const * const filts, float const * const biases, float const * const in, float * const out ) {
  __shared__ float in_smem[%(threadIdx.x_line_x_tile_dim)*%(t_tile_sz)];
  int32_t const blk_filt_ix_sz = %(threadIdx.x_out_chan_tile_dim)*%(t_tile_sz);
  __shared__ float filts_smem[blk_filt_ix_sz];
  float out_tile[%(t_tile_sz)*%(t_tile_sz)] = {0}; // tile of output for this thread to compute, stored in registers
  // reg. buffers for one strip each from in and filts of %(t_tile_sz) elements, for the same filts_ix_out_chan_elem
  float filts_strip[%(t_tile_sz)]; // across output chans (stride is blk_filt_ix_sz )
  float in_strip[%(t_tile_sz)]; // across patches (approx square block in x/y space, favoring x if sqrt() not integer)
  int32_t const blk_filt_ix_base = %(blockIdx.x_out_chan_blk)*blk_filt_ix_sz;

  int32_t const blk_patch_ix_sz = %(threadIdx.x_line_x_tile_dim)*%(t_tile_sz);
  //int32_t const blk_in_base_ix = %(blockIdx.x_img)*%(in_ix_img_sz) + %(blockIdx.x_y)*%(in_ix_y_sz);
  // TODO: start here ....
  // iteratate over filter elements
  int32_t filts_off = blk_filt_ix_base;
  for( int32_t filts_ix_out_chan_elem = 0; filts_ix_out_chan_elem != (%(filts_xp_ix_sz) / %(filts_xp_ix_x_sz));
       ++filts_ix_out_chan_elem ) {
    __syncthreads();
    for( int32_t i = 0; i != %(out_chan_smem_load_iter); ++i ) {
      int32_t const t_smem_filt_ix = threadIdx.x+blockDim.x*i;
      if( t_smem_filt_ix < blk_filt_ix_sz ) { 
#ifdef NO_IOX // by default, we don't ever disable this, since it's seems about as good as it can be already
	//filts_smem[threadIdx.x] = threadIdx.x;
	filts_smem[t_smem_filt_ix] = filts[threadIdx.x];
#else
	filts_smem[t_smem_filt_ix] = filts[filts_off+t_smem_filt_ix];
#endif
      }
    }
    for( int32_t i = 0; i != %(patch_smem_load_iter); ++i ) {
      int32_t const t_smem_line_x = threadIdx.x + blockDim.x*i;
      if( t_smem_line_x < blk_patch_ix_sz ) { 
#ifdef NO_IO2
	//float v = threadIdx.x;
	//float v = in[threadIdx.x];
	float v = in[t_smem_line_x];
#else
	%(get_in);
#endif
	in_smem[t_smem_line_x] = v;
      }
    }
    filts_off += %(filts_xp_ix_x_sz);
    __syncthreads();
#ifdef NO_IO2
    %(t_tile_dummy_loads);
#else
    %(t_tile_loads);
#endif
    // (2) do %(t_tile_sz)^2 fmas into out_tile
    %(t_tile_fmas);
  }

  // load per-block biases into smem
  __syncthreads();
    for( int32_t i = 0; i != %(out_chan_smem_load_iter); ++i ) {
      int32_t const t_smem_bias_ix = threadIdx.x+blockDim.x*i;
      if( t_smem_bias_ix < blk_filt_ix_sz ) { 
	int32_t const ocix_base = %(blockIdx.x_out_chan_blk)*blk_filt_ix_sz;
	int32_t const load_reg = t_smem_bias_ix / %(threadIdx.x_out_chan_tile_dim);
	int32_t const load_tile = t_smem_bias_ix %% %(threadIdx.x_out_chan_tile_dim);
	int32_t const ocix = ocix_base + load_tile*%(t_tile_sz) + load_reg;
	if( ocix < %(out_ix_chan_dim) ) { filts_smem[t_smem_bias_ix] = biases[ ocix ]; }
      }
  }
  __syncthreads();
  // load biases into filts_strip
  %(t_tile_loads);

  // add bias to each elem of out_tile[] and store the results to out[]
#ifdef NO_IO2
  %(t_tile_dummy_stores);
#else
  %(t_tile_stores);
#endif
}

