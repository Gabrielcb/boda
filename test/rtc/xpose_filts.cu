#include "hip/hip_runtime.h"
extern "C"  __global__ void %(cu_func_name)( float const * const in, float * const out ) {
  int32_t const filts_ix = blockDim.x * blockIdx.x + threadIdx.x;
  if( filts_ix >= %(filts_ix_sz) ) { return; }
  int32_t const fioc = %(filts_ix_out_chan);
  
  int32_t const filts_xp_ix  = 
    %(fioc_out_chan_blk)*%(filts_xp_ix_out_chan_blk_sz) +
    %(fioc_out_chan_reg)*%(filts_xp_ix_out_chan_reg_sz) +
    %(fioc_out_chan_tile)*%(filts_xp_ix_out_chan_tile_sz) +
    %(filts_ix_in_chan)*%(filts_xp_ix_in_chan_sz) +
    %(filts_ix_y)*%(filts_xp_ix_y_sz) +
    %(filts_ix_x)*%(filts_xp_ix_x_sz);
  out[filts_xp_ix] = in[filts_ix];
}

