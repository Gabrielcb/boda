#include "hip/hip_runtime.h"
extern "C"  __global__ void %(cu_func_name)( float const * const in, float * const out ) {
  int32_t const filts_ix = blockDim.x * blockIdx.x + threadIdx.x;
  if( filts_ix >= %(filts_ix_sz) ) { return; }
  int32_t const fioc = %(filts_ix_out_chan);
  float val = 0.0f;  
  int32_t const filts_xp_ix  = 
    %(fioc_out_chan_blk)*%(filts_xp_ix_out_chan_blk_sz) +
    %(fioc_out_chan_reg)*%(filts_xp_ix_out_chan_reg_sz) +
    %(fioc_out_chan_tile)*%(filts_xp_ix_out_chan_tile_sz) +
    %(filts_ix_in_chan)*%(filts_xp_ix_in_chan_sz) +
    %(filts_ix_y)*%(filts_xp_ix_y_sz) +
    %(filts_ix_x)*%(filts_xp_ix_x_sz);
#if 1
  val = in[filts_ix];
#else
  if( %(filts_ix_in_chan) == 0 ) {
//    if( (%(filts_ix_x) == 5) && (%(filts_ix_y) == 5) ) 
    {
      val = %(filts_ix_x)*100 + %(filts_ix_y);
    }
  }
#endif
  out[filts_xp_ix] = val;
}

