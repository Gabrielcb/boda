#include "hip/hip_runtime.h"
extern "C"  __global__ void %(cu_func_name)( float const * const in, float * const out ) {
  uint32_t const out_ix = blockDim.x * blockIdx.x + threadIdx.x;
  if( out_ix >= %(out_ix_sz) ) { return; }
  float out_v = 0.0f;
  uint32_t const in_ix = %(out_ix_img) * %(in_ix_img_sz) + %(out_ix_chan) * %(in_ix_chan_sz) + 
    %(out_ix_y)*%(in_ix_y_sz)*%(stride) + %(out_ix_x)*%(in_ix_x_sz)*%(stride);
  %(ops);
  out[out_ix] = out_v;
}
