#include "hip/hip_runtime.h"
// 256 tbp
// each thread: computes 8x8 block of out
// loop over k dim
extern "C"  __global__ void %(cu_func_name)( float const * const filts, float const * const biases, float const * const in, float * const out ) {
  __shared__ float in_smem[%(threadIdx.x_patch_tile_dim)*%(t_tile_sz)];
  __shared__ float filts_smem[%(threadIdx.x_out_chan_tile_dim)*%(t_tile_sz)];
  float out_tile[%(t_tile_sz)*%(t_tile_sz)] = {0}; // tile of output for this thread to compute, stored in registers
  // reg. buffers for one strip each from in and filts of %(t_tile_sz) elements, for the same filts_ix_out_chan_elem
  float filts_strip[%(t_tile_sz)]; // across output chans (stride is %(filts_ix_out_chan_sz) )
  float in_strip[%(t_tile_sz)]; // across patches (approx square block in x/y space, favoring x if sqrt() not integer)
  uint32_t const blk_filt_ix_sz = %(threadIdx.x_out_chan_tile_dim)*%(t_tile_sz);
  uint32_t const blk_filt_ix_base = %(blockIdx.x_out_chan_blk)*blk_filt_ix_sz;

  uint32_t const blk_patch_ix_sz = %(threadIdx.x_patch_tile_dim)*%(t_tile_sz);
  uint32_t const blk_patch_ix_base = %(blockIdx.x_patch_blk)*blk_patch_ix_sz;

  // iteratate over filter elements
  for( uint32_t filts_ix_out_chan_elem = 0; filts_ix_out_chan_elem != %(filts_ix_out_chan_sz); ++filts_ix_out_chan_elem ) {
    // (1) load %(t_tile_sz) elements from in and filts    
    __syncthreads();
    if( threadIdx.x < blk_filt_ix_sz ) { 
      filts_smem[threadIdx.x] = filts[(blk_filt_ix_base+threadIdx.x)*%(filts_ix_out_chan_sz) + filts_ix_out_chan_elem];
    }
    for( uint32_t i = 0; i != %(patch_smem_load_iter); ++i ) {
      if( (threadIdx.x+blockDim.x*i) < blk_patch_ix_sz ) { 
	uint32_t const t_smem_patch_ix = (blk_patch_ix_base+threadIdx.x+blockDim.x*i);
	%(get_in);
#if 0
	int64_t iv = v;
	if( iv > 200 ) { iv = 200; }
	if( iv < -200 ) { iv = -200; }
	v = iv;
#endif
	in_smem[threadIdx.x+blockDim.x*i] = v;
      }
    }
    __syncthreads();
    %(t_tile_loads);
    // (2) do %(t_tile_sz)^2 fmas into out_tile
    %(t_tile_fmas);
  }
  // add bias to each elem of out_tile[] and store the results to out[]
  %(t_tile_stores);
}

