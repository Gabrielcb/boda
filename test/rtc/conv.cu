#include "hip/hip_runtime.h"
// 256 tbp
// each thread: computes 8x8 block of out
// loop over k dim
extern "C"  __global__ void %(cu_func_name)( float const * const filts, float const * const biases, float const * const in, float * const out ) {
  __shared__ float in_smem[%(threadIdx.x_patch_tile_dim)*%(t_tile_sz)];
  __shared__ float filts_smem[%(threadIdx.x_out_chan_tile_dim)*%(t_tile_sz)];
  float out_tile[%(t_tile_sz)*%(t_tile_sz)] = {0}; // tile of output for this thread to compute, stored in registers
  // reg. buffers for one strip each from in and filts of %(t_tile_sz) elements, for the same filts_ix_out_chan_elem
  float filts_strip[%(t_tile_sz)]; // across output chans (stride is %(filts_ix_out_chan_sz) )
  float in_strip[%(t_tile_sz)]; // across patches (approx square block in x/y space, favoring x if sqrt() not integer)
  uint32_t const blk_filt_ix_sz = %(threadIdx.x_out_chan_tile_dim)*%(t_tile_sz);
  uint32_t const blk_filt_ix_base = %(blockIdx.x_out_chan_blk)*blk_filt_ix_sz;

  uint32_t const blk_patch_ix_sz = %(threadIdx.x_patch_tile_dim)*%(t_tile_sz);
  uint32_t const blk_patch_ix_base = %(blockIdx.x_patch_blk)*blk_patch_ix_sz;

  // iteratate over filter elements
  for( uint32_t filts_ix_out_chan_elem = 0; filts_ix_out_chan_elem != %(filts_ix_out_chan_sz); ++filts_ix_out_chan_elem ) {
    // (1) load %(t_tile_sz) elements from in and filts    
    __syncthreads();
    if( threadIdx.x < blk_filt_ix_sz ) { 
      filts_smem[threadIdx.x] = filts[(blk_filt_ix_base+threadIdx.x)*%(filts_ix_out_chan_sz) + filts_ix_out_chan_elem];
    }
    if( threadIdx.x < blk_patch_ix_sz ) { 
      uint32_t const t_smem_patch_ix = (blk_patch_ix_base+threadIdx.x);
      in_smem[threadIdx.x] = in[%(t_smem_patch_ix_img)*%(in_ix_img_sz) +
				%(filts_ix_out_chan_elem_in_chan)*%(in_ix_chan_sz) +
				(%(t_smem_patch_ix_y)*%(stride)+%(filts_ix_out_chan_elem_y))*%(in_ix_y_sz) +
				(%(t_smem_patch_ix_x)*%(stride)+%(filts_ix_out_chan_elem_x))*%(in_ix_x_sz)];
    }
    if( 0 && threadIdx.x+blockDim.x < blk_patch_ix_sz ) { 
      uint32_t const t_smem_patch_ix = (blk_patch_ix_base+threadIdx.x+blockDim.x);
      in_smem[threadIdx.x+blockDim.x] = in[%(t_smem_patch_ix_img)*%(in_ix_img_sz) +
					   %(filts_ix_out_chan_elem_in_chan)*%(in_ix_chan_sz) +
					   (%(t_smem_patch_ix_y)*%(stride)+%(filts_ix_out_chan_elem_y))*%(in_ix_y_sz) +
					   (%(t_smem_patch_ix_x)*%(stride)+%(filts_ix_out_chan_elem_x))*%(in_ix_x_sz)];
    }
    __syncthreads();
    %(t_tile_loads);
    // (2) do %(t_tile_sz)^2 fmas into out_tile
    %(t_tile_fmas);
  }
  // add bias to each elem of out_tile[] and store the results to out[]
  %(t_tile_stores);
}

